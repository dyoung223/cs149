#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result

__global__ void upsweep_phase(int two_dplus1, int two_d, int N, int* result){

    int index = two_dplus1*(blockIdx.x * blockDim.x + threadIdx.x);
    int taskOutputIndex = index + two_dplus1 - 1;
    int taskInputIndex = index + two_dplus - 1; 
    if(taskOutputIndex < N && taskInputIndex < N){
        result[taskOutputIndex] += result[taskInputIndex];
    }
}

__global__ void downsweep_phase(int two_dplus1, int two_d, int N, int* result){

    int index = two_dplus1*(blockIdx.x * blockDim.x + threadIdx.x);
    
    int taskOutputIndex = index + two_dplus1 - 1;
    int taskInputIndex = index + two_dplus - 1; 
    if(taskOutputIndex < N && taskInputIndex < N){
        int t = result[taskInputIndex];
        result[taskInputIndex] = result[taskOutputIndex];
        result[taskOutputIndex] += t;
    }
}
void exclusive_scan(int* input, int N, int* result)
{

    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep in
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.
    
    int rounded_length = nextPow2(N);
    const int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    // There is a case where you need to combine blocks information and don't need all the blocks turned on 
    /*
    if(N >= 256){
        for(int two_d = 1; two_d <= N/2; two_d*=2){
            int num_threads = THREADS_PER_BLOCK/two_d;
            int two_dplus1 = 2*two_d;
            upsweep_phase<<<blocks, num_threads>>>(two_dplus1, two_d, N, input, result);
        }
     }else{
        for(int two_d = 1; two_d <= N/2; two_d*=2){
            int num_threads = N/two_d;
            int two_dplus1 = 2*two_d;
            upsweep_phase<<<blocks, num_threads>>>(two_dplus1, two_d, N, input, result);
        }
     }
     */
     int two_d = 1;
     int two_dplus1 = 2*two_d;
     int num_ops = rounded_length/2;

     //handles the case where you have multiple threads
     while(num_ops >= THREADS_PER_BLOCK){
        int num_blocks = num_ops/THREADS_PER_BLOCK;
        upsweep_phase<<<num_blocks, THREADS_PER_BLOCK>>>(two_dplus1, two_d, rounded_length, result);
        two_d *= 2;
        two_dplus1 = 2*two_d;
        num_ops /= 2;
     }
     while(num_ops > 0}{
        int num_threads = num_ops;
        upsweep_phase<<<1, num_threads>>>(two_dplus1, two_d, rounded_length, result);
        two_d *= 2;
        two_dplus1 = 2*two_d;
        num_ops /= 2;
     }

    //Need to copy the device result back to CPU to change the last index????
    //int* resultarray = new int[N];
    //hipMemcpy(resultarray, device_result, N * sizeof(int), hipMemcpyDeviceToHost);
    //result_array[N-1] = 0;
    //hipMemcpy(device_result, resultarray, N * sizeof(int), hipMemcpyDeviceToHost);

    //cudaMemSet
    hipMemset(device_result+(rounded_length-1)*sizeof(int), 0, sizeof(int));  

    two_d = rounded_length/2;
    two_dplus1 = 2*two_d;
    num_ops = 1;
    
    while(num_ops < THREADS_PER_BLOCK){
        num_threads = num_ops;
        downsweep_phase<<<1, num_threads>>>(two_dplus1, two_d, rounded_length, result);
        two_d /= 2;
        two_dplus1 = 2*two_d;
        num_ops *= 2;
    }
    while(two_d >= 1){
        num_blocks = num_ops/THREADS_PER_BLOCK;
        downsweep_phase<<<num_blocks, THREADS_PER_BLOCK>>>(two_dplus1, two_d, rounded_length, result);

    }
    /*
    for(int two_d = N/2; two_d >= 1; two_d/=2){
        int two_dplus1 = 2*two_d;
        downsweep_phase<<<blocks, THREADS_PER_BLOCK>>>(two_dplus1, two_d, N, input, result);
        
    }
    */


}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


__global__ void create_mask(int* input, int N, int* output){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < N - 1){
        output[index] = input[index] == input[index + 1] ? 1 : 0;
    }
}
__global__ void assign_index(int* scan, int* mask, int N, int* output){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < N - 1){
        if(mask[index] == 1){
            output[scan[index]] = index;
        }
    }
}
_

// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.
    int rounded_length = nextPow2(length);
    int *mask;
    int *scan;
    const int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    if (rounded_length < THREADS_PER_BLOCK){
        create_mask<<<1, rounded_length>>>(device_input, rounded_length, mask);
        exclusive_scan(mask, rounded_length, scan);
        assign_index<<<1, rounded_length>>>(scan, mask, rounded_length, device_output);
    else{
        create_mask<<<blocks, THREADS_PER_BLOCK>>>(device_input, rounded_length, mask);
        exclusive_scan(mask, rounded_length, scan);
        assign_index<<<blocks, THREADS_PER_BLOCK>>>(scan, mask, rounded_length, device_output);
    }
    

    return mask[rounded_length-1]; 
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
